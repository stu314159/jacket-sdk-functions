#include "hip/hip_runtime.h"
#include "jacketSDK.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/device_functions.h"
#include <iostream>
#define TPB 16


__global__ void chan_cav_D2Q9_LBGK_ts(float * fOut, float * fIn, int * inl, 
				 int * onl, int * snl, float * ux_p,
				 const float omega, const int Nx,
				 const int Ny){

  int X=threadIdx.x+blockIdx.x*blockDim.x;
  int Y=threadIdx.y+blockIdx.y*blockDim.y;
  if((X<Nx)&&(Y<Ny)){
    int tid=X+Y*Nx;
    int nnodes=Nx*Ny;
    float fi1,fi2,fi3,fi4,fi5,fi6,fi7,fi8,fi9;
    float fe_tmp;
    float fo1,fo2,fo3,fo4,fo5,fo6,fo7,fo8,fo9;
    float w, cu;

    //get the density data for the lattice point.
    fi1=fIn[tid];
    fi2=fIn[nnodes+tid];
    fi3=fIn[2*nnodes+tid];
    fi4=fIn[3*nnodes+tid];
    fi5=fIn[4*nnodes+tid];
    fi6=fIn[5*nnodes+tid];
    fi7=fIn[6*nnodes+tid];
    fi8=fIn[7*nnodes+tid];
    fi9=fIn[8*nnodes+tid];

    //compute rho
    float rho = fi1+fi2+fi3+fi4+fi5+fi6+fi7+fi8+fi9;
    

    //compute velocity
    float ux = (1/rho)*(fi2+fi6+fi9 - (fi7+fi4+fi8));
    float uy = (1/rho)*(fi6+fi3+fi7 - (fi8+fi5+fi9));
   


    //insert BC code here...

    //if tid is an inlet node, set inlet Macroscopic and microscopic BC
    if((inl[tid]==1)||(onl[tid]==1)){
      float dx=ux_p[tid]-ux;
      float dy = -uy; //since for this uy_p==0

      //speed 2
      w=1./9.;
      cu = 3.*(dx);
      fi2+=w*rho*cu;

      //speed 3
      cu=3.*dy;
      fi3+=w*rho*cu;

      //speed 4
      cu=3.*(-dx);
      fi4+=w*rho*cu;

      //speed 5
      cu = 3.*(-dy);
      fi5+=w*rho*cu;

      //speed 6
      w=1./36.;
      cu=3.*(dx+dy);
      fi6+=w*rho*cu;

      //speed 7
      cu=3.*(-dx+dy);
      fi7+=w*rho*cu;

      //speed 8
      cu=3.*(-dx-dy);
      fi8+=w*rho*cu;

      //speed 9
      cu=3.*(dx-dy);
      fi9+=w*rho*cu;
    }

    //compute feq and collide...do it one velocity at a time.
    //speed 1
    w = 4./9.;
    cu = 0.;
    fe_tmp = w*rho*(1.+cu+0.5*(cu*cu)-1.5*(ux*ux + uy*uy));
    fo1 = fi1-omega*(fi1-fe_tmp);

    //speed 2
    w = 1./9.;
    cu = 3.0*ux;
    fe_tmp = w*rho*(1.+cu+0.5*(cu*cu)-1.5*(ux*ux + uy*uy));
    fo2 = fi2-omega*(fi2-fe_tmp);

    //speed 3
    cu = 3.0*uy;
    fe_tmp = w*rho*(1.+cu+0.5*(cu*cu)-1.5*(ux*ux + uy*uy));
    fo3 = fi3-omega*(fi3-fe_tmp);

    //speed 4
    cu = -3.0*ux;
    fe_tmp = w*rho*(1.+cu+0.5*(cu*cu)-1.5*(ux*ux + uy*uy));
    fo4=fi4-omega*(fi4-fe_tmp);

    //speed 5
    cu = -3.0*uy;
    fe_tmp = w*rho*(1.+cu+0.5*(cu*cu)-1.5*(ux*ux + uy*uy));
    fo5=fi5-omega*(fi5-fe_tmp);

    //speed 6
    w = 1./36.;
    cu = 3.0*(ux+uy);
    fe_tmp = w*rho*(1.+cu+0.5*(cu*cu)-1.5*(ux*ux + uy*uy));
    fo6 = fi6-omega*(fi6-fe_tmp);

    //speed 7
    cu = 3.0*(-ux+uy);
    fe_tmp = w*rho*(1.+cu+0.5*(cu*cu)-1.5*(ux*ux + uy*uy));
    fo7=fi7-omega*(fi7-fe_tmp);

    //speed 8
    cu = 3.0*(-ux-uy);
    fe_tmp = w*rho*(1.+cu+0.5*(cu*cu)-1.5*(ux*ux + uy*uy));
    fo8=fi8-omega*(fi8-fe_tmp);

    //speed 9
    cu= 3.0*(ux-uy);
    fe_tmp = w*rho*(1.+cu+0.5*(cu*cu)-1.5*(ux*ux + uy*uy));
    fo9=fi9-omega*(fi9-fe_tmp);

    if(snl[tid]==1){//solid node

      fo1=fi1;
      fo2=fi4; fo4=fi2;
      fo3=fi5; fo5=fi3;
      fo6=fi8; fo8=fi6;
      fo7=fi9; fo9=fi7;
      ux = 0.; uy = 0.;
    }

    // stream the result...
    //compute the local stream vector...	
    int x;
    int y;
    int yn;
    int ys;
    int xe;
    int xw;
   
    //int dir; 
    int dof_num; //int f_num;
    x = tid%Nx+1;
    y = ((tid+1)-x+1)/Nx + 1; 

    yn = y%Ny+1;
    xe = x%Nx+1;

    if(y==1){
      ys = Ny;
    }else{
      ys = y-1;
    }
    if(x==1){
      xw=Nx;
    }else{
      xw=x-1;
    }

    dof_num = Nx*(y-1)+x;
    fOut[dof_num-1]=fo1;

    dof_num=Nx*(y-1)+xe;
    fOut[nnodes+dof_num-1]=fo2;

    dof_num=Nx*(yn-1)+x;
    fOut[2*nnodes+dof_num-1]=fo3;

    dof_num=Nx*(y-1)+xw;
    fOut[3*nnodes+dof_num-1]=fo4;

    dof_num=Nx*(ys-1)+x;
    fOut[4*nnodes+dof_num-1]=fo5;

    dof_num=Nx*(yn-1)+xe;
    fOut[5*nnodes+dof_num-1]=fo6;

    dof_num=Nx*(yn-1)+xw;
    fOut[6*nnodes+dof_num-1]=fo7;

    dof_num=Nx*(ys-1)+xw;
    fOut[7*nnodes+dof_num-1]=fo8;

    dof_num=Nx*(ys-1)+xe;
    fOut[8*nnodes+dof_num-1]=fo9;

  }

}


err_t jktFunction(int nlhs,mxArray * plhs[],int nrhs,mxArray * prhs[]){


  if(nrhs!=9)
    return err("Usage: ChanCav_D2Q9_LBGK_ts(fOut,fIn,inl,onl,snl,ux_p,omega,Nx,Ny)");

  mxArray * m_fOut=prhs[0];
  mxArray * m_fIn=prhs[1];
  mxArray * m_inl=prhs[2];
  mxArray * m_onl=prhs[3];
  mxArray * m_snl=prhs[4];
  mxArray * m_ux_p=prhs[5];
  float omega=mxGetScalar(prhs[6]);
  int Nx = mxGetScalar(prhs[7]);
  int Ny =mxGetScalar(prhs[8]);

  float * fOut_d;
  float * fIn_d;
  int * inl_d;
  int * onl_d;
  int * snl_d;
  float * ux_p_d;

  jkt_mem((void**)&fOut_d,m_fOut);
  jkt_mem((void**)&fIn_d,m_fIn);
  jkt_mem((void**)&inl_d,m_inl);
  jkt_mem((void**)&onl_d,m_onl);
  jkt_mem((void**)&snl_d,m_snl);
  jkt_mem((void**)&ux_p_d,m_ux_p);

  dim3 BLOCKS(TPB,TPB,1);
  dim3 GRIDS((Nx+TPB-1)/TPB,(Ny+TPB-1)/TPB,1);

  chan_cav_D2Q9_LBGK_ts<<<GRIDS,BLOCKS>>>(fOut_d,fIn_d,inl_d,onl_d,
					  snl_d,ux_p_d,omega,Nx,Ny);

  return errNone;



}
